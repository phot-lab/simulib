/**
 * Copyright (c) 2022 Beijing Jiaotong University
 * OpticaLab is licensed under [Open Source License].
 * You can use this software according to the terms and conditions of the [Open Source License].
 * You may obtain a copy of [Open Source License] at: [https://open.source.license/]
 *
 * THIS SOFTWARE IS PROVIDED ON AN "AS IS" BASIS, WITHOUT WARRANTIES OF ANY KIND,
 * EITHER EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO NON-INFRINGEMENT,
 * MERCHANTABILITY OR FIT FOR A PARTICULAR PURPOSE.
 *
 * See the [Open Source License] for more details.
 */
/**
 * Author: Chunyu Li
 * Created: 2022/3/31
 * Supported by: National Key Research and Development Program of China
 */

#include <Internal>
#include <complex>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <iostream>

using namespace std;

#define IDX(i, j, ld) (((j) * (ld)) + (i))

static void cuFFT(complex<double> *data, int rows, int cols) {
    hipfftHandle plan;
    hipfftDoubleComplex *dataDev;
    HANDLE_ERROR(hipMalloc((void **) &dataDev, sizeof(hipfftDoubleComplex) * rows * cols));
    HANDLE_ERROR(hipMemcpy(dataDev, data, sizeof(hipfftDoubleComplex) * rows * cols, hipMemcpyHostToDevice));

    if (hipfftPlan1d(&plan, rows, HIPFFT_Z2Z, cols) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: Plan creation failed");
        return;
    }

    // Notes: Identical pointers to input and output arrays implies in-place transformation
    if (hipfftExecZ2Z(plan, dataDev, dataDev, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
        return;
    }

    hipMemcpy(data, dataDev, sizeof(hipfftDoubleComplex) * rows * cols, hipMemcpyDeviceToHost);
}

static void cuScale(complex<double> *data, complex<double> alpha, int rows, int cols) {
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        cout << "CUBLAS initialization failed" << endl;
        return;
    }
    hipfftDoubleComplex *dataDev;
    hipMalloc(&dataDev, sizeof(hipfftDoubleComplex) * rows * cols);

    hipblasSetMatrix(rows, cols, sizeof(hipfftDoubleComplex), data, rows, dataDev, rows);
    hipfftDoubleComplex temp;
    temp.x = alpha.real();
    temp.y = alpha.imag();
    hipblasZscal(handle, rows * cols, &temp, dataDev, 1);
    hipblasGetMatrix(rows, cols, sizeof(hipfftDoubleComplex), dataDev, rows, data, rows);

    hipFree(dataDev);
    hipblasDestroy(handle);
}

static void cuIFFT(complex<double> *data, int rows, int cols) {
    hipfftHandle plan;
    hipfftDoubleComplex *dataDev;
    HANDLE_ERROR(hipMalloc((void **) &dataDev, sizeof(hipfftDoubleComplex) * rows * cols));
    HANDLE_ERROR(hipMemcpy(dataDev, data, sizeof(hipfftDoubleComplex) * rows * cols, hipMemcpyHostToDevice));

    if (hipfftPlan1d(&plan, rows, HIPFFT_Z2Z, cols) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: Plan creation failed");
        return;
    }

    // Notes: Identical pointers to input and output arrays implies in-place transformation
    if (hipfftExecZ2Z(plan, dataDev, dataDev, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
        return;
    }

    hipMemcpy(data, dataDev, sizeof(hipfftDoubleComplex) * rows * cols, hipMemcpyDeviceToHost);
    hipfftDestroy(plan);
    hipFree(dataDev);
    complex<double> scalar((double) 1 / (rows), 0);
    cuScale(data, scalar, rows, cols);
}

namespace SimuLib {

namespace GPU{

VectorXcd fft(const VectorXcd &in) {
    VectorXcd out = in;
    cuFFT(out.data(), (int) out.rows(), (int) out.cols());
    return out;
}

VectorXcd ifft(const VectorXcd &in) {
    VectorXcd out = in;
    cuIFFT(out.data(), (int) out.rows(), (int) out.cols());
    return out;
}

MatrixXcd fftCol(const MatrixXcd &in) {
    MatrixXcd out(in.rows(), in.cols());
    for (Index i = 0; i < in.cols(); ++i) {
        out.col(i) = fft(in.col(i));
    }
    return out;
}

MatrixXcd ifftCol(const MatrixXcd &in) {
    MatrixXcd out(in.rows(), in.cols());
    for (Index i = 0; i < in.cols(); ++i) {
        out.col(i) = ifft(in.col(i));
    }
    return out;
}

}

}  // namespace SimuLib
