/**
 * Copyright (c) 2022 Beijing Jiaotong University
 * OpticaLab is licensed under [Open Source License].
 * You can use this software according to the terms and conditions of the [Open Source License].
 * You may obtain a copy of [Open Source License] at: [https://open.source.license/]
 *
 * THIS SOFTWARE IS PROVIDED ON AN "AS IS" BASIS, WITHOUT WARRANTIES OF ANY KIND,
 * EITHER EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO NON-INFRINGEMENT,
 * MERCHANTABILITY OR FIT FOR A PARTICULAR PURPOSE.
 *
 * See the [Open Source License] for more details.
 */
/**
 * Author: Chunyu Li
 * Created: 2022/4/1
 * Supported by: National Key Research and Development Program of China
 */

#include <cstdio>
#include <hipblaslt.h>
#include <cublasXt.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

void handleError(hipError_t err, const char *file, int line) {

    // CUDA error handling from the "CUDA by example" book
    if (err != hipSuccess) {
        std::cerr << hipGetErrorString(err) << std::endl;
        std::cerr << "File Name: " << file << std::endl;  // File name
        std::cerr << "Code Line: " << line << std::endl;  // Code line
        exit(EXIT_FAILURE);
    }
}

static const char *cublasGetStatusString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
    }

    return "<unknown>";
}

void handleStatus(hipblasStatus_t status, const char *file, int line) {

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << cublasGetStatusString(status) << std::endl;
        std::cerr << "File Name: " << file << std::endl;  // File name
        std::cerr << "Code Line: " << line << std::endl;  // Code line
        exit(EXIT_FAILURE);
    }
}
